#include "hip/hip_runtime.h"
/* Boost check_thrust.cu test file
 
 Copyright 2009 Karsten Ahnert
 Copyright 2009 Mario Mulansky
 
 This file tests the use of the euler stepper
  
 Distributed under the Boost Software License, Version 1.0.
 (See accompanying file LICENSE_1_0.txt or
 copy at http://www.boost.org/LICENSE_1_0.txt)
*/

//#include <boost/test/unit_test.hpp>

#include <boost/numeric/odeint/stepper/euler.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

#include <thrust/device_vector.h>
#include <thrust/fill.h>

using namespace boost::numeric::odeint;

typedef float base_type;
// typedef thrust::device_vector< base_type > state_type;
typedef thrust::host_vector< base_type > state_type;

void constant_system( const state_type &x , state_type &dxdt , base_type t )
{
	thrust::fill( dxdt.begin() , dxdt.end() , static_cast<base_type>(1.0) );
}

const base_type eps = 1.0e-7;


template< class Stepper , class System >
void check_stepper_concept( Stepper &stepper , System system , typename Stepper::state_type &x )
{
    typedef Stepper stepper_type;
    typedef typename stepper_type::state_type container_type;
    typedef typename stepper_type::order_type order_type;
    typedef typename stepper_type::time_type time_type;

    stepper.do_step( system , x , 0.0 , 0.1 );
    base_type xval = *boost::begin( x );
    if( fabs( xval - 0.1 ) < eps )
    	std::clog << "TEST PASSED" << std::endl;
    else
    	std::clog << "TEST FAILED" << std::endl;
}

void test_euler_with_thrust( void )
{
	state_type x(1);
	thrust::fill( x.begin() , x.end() , static_cast<base_type>(0.0) );
	euler< state_type , base_type , state_type , base_type , thrust_algebra , thrust_operations > euler;
	check_stepper_concept( euler , constant_system , x );


}

/*test_suite* init_unit_test_suite( int argc, char* argv[] )
{
    test_suite *test = BOOST_TEST_SUITE("check stepper with thrust");

    test->add( BOOST_TEST_CASE( &test_euler_with_thrust ) );

    return test;
}*/

int main() {
	test_euler_with_thrust();
}
