#include "hip/hip_runtime.h"
/*
 * phase_oscillator_ensemble.cu
 *
 * The example how the phase_oscillator ensemble can be implemented using CUDA and thrust
 *
 *  Created on: July 15, 2011
 *      Author: karsten
 */


#include <iostream>
#include <fstream>
#include <cmath>
#include <utility>

#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

#include <boost/timer.hpp>
#include <boost/random/cauchy_distribution.hpp>

using namespace std;

using namespace boost::numeric::odeint;

/*
 * Sorry for that dirty hack, but nvcc has large problems with boost::random.
 *
 * Nevertheless we need the cauchy distribution from boost::random, and therefore
 * we need a generator. Here it is:
 */
struct drand48_generator
{
    typedef double result_type;
    result_type operator()( void ) const { return drand48(); }
    result_type min( void ) const { return 0.0; }
    result_type max( void ) const { return 1.0; }
};

//[ thrust_phase_ensemble_state_type
//change this to float if your device does not support double computation
typedef double value_type;

//change this to host_vector< ... > of you want to run on CPU
typedef thrust::device_vector< value_type > state_type;
// typedef thrust::host_vector< value_type > state_type;
//]


//[ thrust_phase_ensemble_mean_field_calculator
struct mean_field_calculator
{
    struct sin_functor : public thrust::unary_function< value_type , value_type >
    {
        __host__ __device__
        value_type operator()( value_type x) const
        {
            return sin( x );
        }
    };

    struct cos_functor : public thrust::unary_function< value_type , value_type >
    {
        __host__ __device__
        value_type operator()( value_type x) const
        {
            return cos( x );
        }
    };

    static std::pair< value_type , value_type > get_mean( const state_type &x )
    {
        //[ thrust_phase_ensemble_sin_sum
        value_type sin_sum = thrust::reduce(
                thrust::make_transform_iterator( x.begin() , sin_functor() ) ,
                thrust::make_transform_iterator( x.end() , sin_functor() ) );
        //]
        value_type cos_sum = thrust::reduce(
                thrust::make_transform_iterator( x.begin() , cos_functor() ) ,
                thrust::make_transform_iterator( x.end() , cos_functor() ) );

        cos_sum /= value_type( x.size() );
        sin_sum /= value_type( x.size() );

        value_type K = sqrt( cos_sum * cos_sum + sin_sum * sin_sum );
        value_type Theta = atan2( sin_sum , cos_sum );

        return std::make_pair( K , Theta );
    }
};
//]



//[ thrust_phase_ensemble_sys_function
class phase_oscillator_ensemble
{

public:

    struct sys_functor
    {
        value_type m_K , m_Theta , m_epsilon;

        sys_functor( value_type K , value_type Theta , value_type epsilon )
        : m_K( K ) , m_Theta( Theta ) , m_epsilon( epsilon ) { }

        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )
        {
            thrust::get<2>(t) = thrust::get<1>(t) + m_epsilon * m_K * sin( m_Theta - thrust::get<0>(t) );
        }
    };

    // ...
    //<-
    phase_oscillator_ensemble( size_t N , value_type g = 1.0 , value_type epsilon = 1.0 )
        : m_omega() , m_N( N ) , m_epsilon( epsilon )
    {
        create_frequencies( g );
    }

    void create_frequencies( value_type g )
    {
        boost::cauchy_distribution< value_type > cauchy( 0.0 , g );
//        boost::variate_generator< boost::mt19937&, boost::cauchy_distribution< value_type > > gen( rng , cauchy );
        drand48_generator d48;
        vector< value_type > omega( m_N );
        for( size_t i=0 ; i<m_N ; ++i )
            omega[i] = cauchy( d48 );
//        generate( omega.begin() , omega.end() , gen );
        m_omega = omega;
    }

    void set_epsilon( value_type epsilon ) { m_epsilon = epsilon; }

    value_type get_epsilon( void ) const { return m_epsilon; }
    //->

    void operator() ( const state_type &x , state_type &dxdt , const value_type dt ) const
    {
        std::pair< value_type , value_type > mean_field = mean_field_calculator::get_mean( x );

        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple( x.begin() , m_omega.begin() , dxdt.begin() ) ),
                thrust::make_zip_iterator( thrust::make_tuple( x.end() , m_omega.end() , dxdt.end()) ) ,
                sys_functor( mean_field.first , mean_field.second , m_epsilon )
                );
    }

    // ...
    //<-
private:

    state_type m_omega;
    const size_t m_N;
    value_type m_epsilon;
    //->
};
//]


//[ thrust_phase_ensemble_observer
struct statistics_observer
{
    value_type m_K_mean;
    size_t m_count;

    statistics_observer( void )
    : m_K_mean( 0.0 ) , m_count( 0 ) { }

    template< class State >
    void operator()( const State &x , value_type t )
    {
        std::pair< value_type , value_type > mean = mean_field_calculator::get_mean( x );
        m_K_mean += mean.first;
        ++m_count;
    }

    value_type get_K_mean( void ) const { return ( m_count != 0 ) ? m_K_mean / value_type( m_count ) : 0.0 ; }

    void reset( void ) { m_K_mean = 0.0; m_count = 0; }
};
//]



// const size_t N = 16384 * 128;
const size_t N = 16384;
const value_type pi = 3.1415926535897932384626433832795029;
const value_type dt = 0.1;
const value_type d_epsilon = 0.1;
const value_type epsilon_min = 0.0;
const value_type epsilon_max = 5.0;
const value_type t_transients = 10.0;
const value_type t_max = 100.0;

int main( int arc , char* argv[] )
{
    // initial conditions on host
    vector< value_type > x_host( N );
    for( size_t i=0 ; i<N ; ++i ) x_host[i] = 2.0 * pi * drand48();

    //[ thrust_phase_ensemble_system_instance
    phase_oscillator_ensemble ensemble( N , 1.0 );
    //]



    boost::timer timer;
    boost::timer timer_local;
    double dopri5_time = 0.0 , rk4_time = 0.0;
    {
        //[thrust_phase_ensemble_define_dopri5
        typedef runge_kutta_dopri5< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper_type;
        //]

        ofstream fout( "phase_ensemble_dopri5.dat" );
        timer.restart();
        for( value_type epsilon = epsilon_min ; epsilon < epsilon_max ; epsilon += d_epsilon )
        {
            ensemble.set_epsilon( epsilon );
            statistics_observer obs;
            state_type x = x_host;

            timer_local.restart();

            // calculate some transients steps
            //[ thrust_phase_ensemble_integration
            size_t steps1 = integrate_const( make_controlled( 1.0e-6 , 1.0e-6 , stepper_type() ) , boost::ref( ensemble ) , x , 0.0 , t_transients , dt );
            //]

            // integrate and compute the statistics
            size_t steps2 = integrate_const( make_dense_output( 1.0e-6 , 1.0e-6 , stepper_type() ) , boost::ref( ensemble ) , x , 0.0 , t_max , dt , boost::ref( obs ) );

            fout << epsilon << "\t" << obs.get_K_mean() << endl;
            cout << "Dopri5 : " << epsilon << "\t" << obs.get_K_mean() << "\t" << timer_local.elapsed() << "\t" << steps1 << "\t" << steps2 << endl;
        }
        dopri5_time = timer.elapsed();
    }



    {
        //[ thrust_phase_ensemble_define_rk4
        typedef runge_kutta4< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper_type;
        //]

        ofstream fout( "phase_ensemble_rk4.dat" );
        timer.restart();
        for( value_type epsilon = epsilon_min ; epsilon < epsilon_max ; epsilon += d_epsilon )
        {
            ensemble.set_epsilon( epsilon );
            statistics_observer obs;
            state_type x = x_host;

            timer_local.restart();

            // calculate some transients steps
            size_t steps1 = integrate_const( stepper_type() , boost::ref( ensemble ) , x , 0.0 , t_transients , dt );

            // integrate and compute the statistics
            size_t steps2 = integrate_const( stepper_type() , boost::ref( ensemble ) , x , 0.0 , t_max , dt , boost::ref( obs ) );
            fout << epsilon << "\t" << obs.get_K_mean() << endl;
            cout << "RK4     : " << epsilon << "\t" << obs.get_K_mean() << "\t" << timer_local.elapsed() << "\t" << steps1 << "\t" << steps2 << endl;
        }
        rk4_time = timer.elapsed();
    }

    cout << "Dopri 5 : " << dopri5_time << " s\n";
    cout << "RK4     : " << rk4_time << "\n";

    return 0;
}
