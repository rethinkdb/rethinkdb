#include "hip/hip_runtime.h"
 /*
 * phase_oscillator_ensemble.cu
 *
 * The example how the phase_oscillator ensemble can be implemented using CUDA and thrust
 *
 *  Created on: July 15, 2011
 *      Author: karsten
 */


#include <iostream>
#include <cmath>
#include <utility>


#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include <boost/numeric/odeint.hpp>

#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_real.hpp>
#include <boost/random/variate_generator.hpp>


using namespace std;
using namespace boost::numeric::odeint;

//change this to float if your device does not support double computation
typedef double value_type;

//change this to host_vector< ... > of you want to run on CPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
// typedef thrust::host_vector< value_type > state_type;
// typedef thrust::host_vector< size_t > index_vector_type;


const value_type sigma = 10.0;
const value_type b = 8.0 / 3.0;


//[ thrust_lorenz_parameters_define_simple_system
struct lorenz_system
{
    struct lorenz_functor
    {
        template< class T >
        __host__ __device__
        void operator()( T t ) const
        {
            // unpack the parameter we want to vary and the Lorenz variables
            value_type R = thrust::get< 3 >( t );
            value_type x = thrust::get< 0 >( t );
            value_type y = thrust::get< 1 >( t );
            value_type z = thrust::get< 2 >( t );
            thrust::get< 4 >( t ) = sigma * ( y - x );
            thrust::get< 5 >( t ) = R * x - y - x * z;
            thrust::get< 6 >( t ) = -b * z + x * y ;

        }
    };

    lorenz_system( size_t N , const state_type &beta )
    : m_N( N ) , m_beta( beta ) { }

    template< class State , class Deriv >
    void operator()(  const State &x , Deriv &dxdt , value_type t ) const
    {
        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple(
                        boost::begin( x ) ,
                        boost::begin( x ) + m_N ,
                        boost::begin( x ) + 2 * m_N ,
                        m_beta.begin() ,
                        boost::begin( dxdt ) ,
                        boost::begin( dxdt ) + m_N ,
                        boost::begin( dxdt ) + 2 * m_N  ) ) ,
                thrust::make_zip_iterator( thrust::make_tuple(
                        boost::begin( x ) + m_N ,
                        boost::begin( x ) + 2 * m_N ,
                        boost::begin( x ) + 3 * m_N ,
                        m_beta.begin() ,
                        boost::begin( dxdt ) + m_N ,
                        boost::begin( dxdt ) + 2 * m_N ,
                        boost::begin( dxdt ) + 3 * m_N  ) ) ,
                lorenz_functor() );
    }
    size_t m_N;
    const state_type &m_beta;
};
//]

struct lorenz_perturbation_system
{
    struct lorenz_perturbation_functor
    {
        template< class T >
        __host__ __device__
        void operator()( T t ) const
        {
            value_type R = thrust::get< 1 >( t );
            value_type x = thrust::get< 0 >( thrust::get< 0 >( t ) );
            value_type y = thrust::get< 1 >( thrust::get< 0 >( t ) );
            value_type z = thrust::get< 2 >( thrust::get< 0 >( t ) );
            value_type dx = thrust::get< 3 >( thrust::get< 0 >( t ) );
            value_type dy = thrust::get< 4 >( thrust::get< 0 >( t ) );
            value_type dz = thrust::get< 5 >( thrust::get< 0 >( t ) );
            thrust::get< 0 >( thrust::get< 2 >( t ) ) = sigma * ( y - x );
            thrust::get< 1 >( thrust::get< 2 >( t ) ) = R * x - y - x * z;
            thrust::get< 2 >( thrust::get< 2 >( t ) ) = -b * z + x * y ;
            thrust::get< 3 >( thrust::get< 2 >( t ) ) = sigma * ( dy - dx );
            thrust::get< 4 >( thrust::get< 2 >( t ) ) = ( R - z ) * dx - dy - x * dz;
            thrust::get< 5 >( thrust::get< 2 >( t ) ) = y * dx + x * dy - b * dz;
        }
    };

    lorenz_perturbation_system( size_t N , const state_type &beta )
    : m_N( N ) , m_beta( beta ) { }

    template< class State , class Deriv >
    void operator()(  const State &x , Deriv &dxdt , value_type t ) const
    {
        thrust::for_each(
                thrust::make_zip_iterator( thrust::make_tuple(
                        thrust::make_zip_iterator( thrust::make_tuple(
                                boost::begin( x ) ,
                                boost::begin( x ) + m_N ,
                                boost::begin( x ) + 2 * m_N ,
                                boost::begin( x ) + 3 * m_N ,
                                boost::begin( x ) + 4 * m_N ,
                                boost::begin( x ) + 5 * m_N ) ) ,
                        m_beta.begin() ,
                        thrust::make_zip_iterator( thrust::make_tuple(
                                boost::begin( dxdt ) ,
                                boost::begin( dxdt ) + m_N ,
                                boost::begin( dxdt ) + 2 * m_N ,
                                boost::begin( dxdt ) + 3 * m_N ,
                                boost::begin( dxdt ) + 4 * m_N ,
                                boost::begin( dxdt ) + 5 * m_N ) )
                ) ) ,
                thrust::make_zip_iterator( thrust::make_tuple(
                        thrust::make_zip_iterator( thrust::make_tuple(
                                boost::begin( x ) + m_N ,
                                boost::begin( x ) + 2 * m_N ,
                                boost::begin( x ) + 3 * m_N ,
                                boost::begin( x ) + 4 * m_N ,
                                boost::begin( x ) + 5 * m_N ,
                                boost::begin( x ) + 6 * m_N ) ) ,
                        m_beta.begin() ,
                        thrust::make_zip_iterator( thrust::make_tuple(
                                boost::begin( dxdt ) + m_N ,
                                boost::begin( dxdt ) + 2 * m_N ,
                                boost::begin( dxdt ) + 3 * m_N ,
                                boost::begin( dxdt ) + 4 * m_N ,
                                boost::begin( dxdt ) + 5 * m_N ,
                                boost::begin( dxdt ) + 6 * m_N  ) )
                ) ) ,
                lorenz_perturbation_functor() );
    }

    size_t m_N;
    const state_type &m_beta;
};

struct lyap_observer
{
    //[thrust_lorenz_parameters_observer_functor
    struct lyap_functor
    {
        template< class T >
        __host__ __device__
        void operator()( T t ) const
        {
            value_type &dx = thrust::get< 0 >( t );
            value_type &dy = thrust::get< 1 >( t );
            value_type &dz = thrust::get< 2 >( t );
            value_type norm = sqrt( dx * dx + dy * dy + dz * dz );
            dx /= norm;
            dy /= norm;
            dz /= norm;
            thrust::get< 3 >( t ) += log( norm );
        }
    };
    //]

    lyap_observer( size_t N , size_t every = 100 )
    : m_N( N ) , m_lyap( N ) , m_every( every ) , m_count( 0 )
    {
        thrust::fill( m_lyap.begin() , m_lyap.end() , 0.0 );
    }

    template< class Lyap >
    void fill_lyap( Lyap &lyap )
    {
        thrust::copy( m_lyap.begin() , m_lyap.end() , lyap.begin() );
        for( size_t i=0 ; i<lyap.size() ; ++i )
            lyap[i] /= m_t_overall;
    }


    template< class State >
    void operator()( State &x , value_type t )
    {
        if( ( m_count != 0 ) && ( ( m_count % m_every ) == 0 ) )
        {
            thrust::for_each(
                    thrust::make_zip_iterator( thrust::make_tuple(
                            boost::begin( x ) + 3 * m_N ,
                            boost::begin( x ) + 4 * m_N ,
                            boost::begin( x ) + 5 * m_N ,
                            m_lyap.begin() ) ) ,
                    thrust::make_zip_iterator( thrust::make_tuple(
                            boost::begin( x ) + 4 * m_N ,
                            boost::begin( x ) + 5 * m_N ,
                            boost::begin( x ) + 6 * m_N ,
                            m_lyap.end() ) ) ,
                    lyap_functor() );
            clog << t << "\n";
        }
        ++m_count;
        m_t_overall = t;
    }

    size_t m_N;
    state_type m_lyap;
    size_t m_every;
    size_t m_count;
    value_type m_t_overall;
};

const size_t N = 1024*2;
const value_type dt = 0.01;


int main( int arc , char* argv[] )
{
    int driver_version , runtime_version;
    hipDriverGetVersion( &driver_version );
    hipRuntimeGetVersion ( &runtime_version );
    cout << driver_version << "\t" << runtime_version << endl;


    //[ thrust_lorenz_parameters_define_beta
    vector< value_type > beta_host( N );
    const value_type beta_min = 0.0 , beta_max = 56.0;
    for( size_t i=0 ; i<N ; ++i )
        beta_host[i] = beta_min + value_type( i ) * ( beta_max - beta_min ) / value_type( N - 1 );

    state_type beta = beta_host;
    //]

    //[ thrust_lorenz_parameters_integration
    state_type x( 6 * N );

    // initialize x,y,z
    thrust::fill( x.begin() , x.begin() + 3 * N , 10.0 );

    // initial dx
    thrust::fill( x.begin() + 3 * N , x.begin() + 4 * N , 1.0 );

    // initialize dy,dz
    thrust::fill( x.begin() + 4 * N , x.end() , 0.0 );


    // create error stepper, can be used with make_controlled or make_dense_output
    typedef runge_kutta_dopri5< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper_type;


    lorenz_system lorenz( N , beta );
    lorenz_perturbation_system lorenz_perturbation( N , beta );
    lyap_observer obs( N , 1 );

    // calculate transients
    integrate_adaptive( make_controlled( 1.0e-6 , 1.0e-6 , stepper_type() ) , lorenz , std::make_pair( x.begin() , x.begin() + 3 * N ) , 0.0 , 10.0 , dt );

    // calculate the Lyapunov exponents -- the main loop
    double t = 0.0;
    while( t < 10000.0 )
    {
        integrate_adaptive( make_controlled( 1.0e-6 , 1.0e-6 , stepper_type() ) , lorenz_perturbation , x , t , t + 1.0 , 0.1 );
        t += 1.0;
        obs( x , t );
    }

    vector< value_type > lyap( N );
    obs.fill_lyap( lyap );

    for( size_t i=0 ; i<N ; ++i )
        cout << beta_host[i] << "\t" << lyap[i] << "\n";
    //]

    return 0;
}
