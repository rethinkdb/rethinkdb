#include "hip/hip_runtime.h"
/*
 * phase_osc_chain.cu
 *
 *  Created on: Apr 1, 2011
 *      Author: mario
 */

/*
 * This example shows how to use odeint on CUDA devices with thrust.
 * Note that we require at least Version 3.2 of the nVidia CUDA SDK
 * and the thrust library should be installed in the CUDA include
 * folder.
 *
 * As example we use a chain of phase oscillators with nearest neighbour
 * coupling, as described in:
 *
 * Avis H. Cohen, Philip J. Holmes and Richard H. Rand:
 * JOURNAL OF MATHEMATICAL BIOLOGY Volume 13, Number 3, 345-369,
 *
 */

#include <iostream>
#include <cmath>

#include <thrust/device_vector.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <boost/numeric/odeint/stepper/runge_kutta4.hpp>
#include <boost/numeric/odeint/integrate/integrate_const.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_algebra.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_operations.hpp>
#include <boost/numeric/odeint/external/thrust/thrust_resize.hpp>

using namespace std;

using namespace boost::numeric::odeint;


//change this to float if your device does not support double computation
typedef double value_type;


//[ thrust_phase_chain_system
//change this to host_vector< ... > if you want to run on CPU
typedef thrust::device_vector< value_type > state_type;
typedef thrust::device_vector< size_t > index_vector_type;
//typedef thrust::host_vector< value_type > state_type;
//typedef thrust::host_vector< size_t > index_vector_type;

//<-
/*
 * This implements the rhs of the dynamical equation:
 * \phi'_0 = \omega_0 + sin( \phi_1 - \phi_0 )
 * \phi'_i  = \omega_i + sin( \phi_i+1 - \phi_i ) + sin( \phi_i - \phi_i-1 )
 * \phi'_N-1 = \omega_N-1 + sin( \phi_N-1 - \phi_N-2 )
 */
//->
class phase_oscillators
{

public:

    struct sys_functor
    {
        template< class Tuple >
        __host__ __device__
        void operator()( Tuple t )  // this functor works on tuples of values
        {
            // first, unpack the tuple into value, neighbors and omega
            const value_type phi = thrust::get<0>(t);
            const value_type phi_left = thrust::get<1>(t);  // left neighbor
            const value_type phi_right = thrust::get<2>(t); // right neighbor
            const value_type omega = thrust::get<3>(t);
            // the dynamical equation
            thrust::get<4>(t) = omega + sin( phi_right - phi ) + sin( phi - phi_left );
        }
    };

    phase_oscillators( const state_type &omega )
        : m_omega( omega ) , m_N( omega.size() ) , m_prev( omega.size() ) , m_next( omega.size() )
    {
        // build indices pointing to left and right neighbours
        thrust::counting_iterator<size_t> c( 0 );
        thrust::copy( c , c+m_N-1 , m_prev.begin()+1 );
        m_prev[0] = 0; // m_prev = { 0 , 0 , 1 , 2 , 3 , ... , N-1 }

        thrust::copy( c+1 , c+m_N , m_next.begin() );
        m_next[m_N-1] = m_N-1; // m_next = { 1 , 2 , 3 , ... , N-1 , N-1 }
    }

    void operator() ( const state_type &x , state_type &dxdt , const value_type dt )
    {
        thrust::for_each(
                thrust::make_zip_iterator(
                        thrust::make_tuple(
                                x.begin() ,
                                thrust::make_permutation_iterator( x.begin() , m_prev.begin() ) ,
                                thrust::make_permutation_iterator( x.begin() , m_next.begin() ) ,
                                m_omega.begin() ,
                                dxdt.begin()
                                ) ),
                thrust::make_zip_iterator(
                        thrust::make_tuple(
                                x.end() ,
                                thrust::make_permutation_iterator( x.begin() , m_prev.end() ) ,
                                thrust::make_permutation_iterator( x.begin() , m_next.end() ) ,
                                m_omega.end() ,
                                dxdt.end()) ) ,
                sys_functor()
                );
    }

private:

    const state_type &m_omega;
    const size_t m_N;
    index_vector_type m_prev;
    index_vector_type m_next;
};
//]

const size_t N = 32768;
const value_type pi = 3.1415926535897932384626433832795029;
const value_type epsilon = 6.0 / ( N * N ); // should be < 8/N^2 to see phase locking
const value_type dt = 0.1;

int main( int arc , char* argv[] )
{
    //[ thrust_phase_chain_integration
    // create initial conditions and omegas on host:
    vector< value_type > x_host( N );
    vector< value_type > omega_host( N );
    for( size_t i=0 ; i<N ; ++i )
    {
        x_host[i] = 2.0 * pi * drand48();
        omega_host[i] = ( N - i ) * epsilon; // decreasing frequencies
    }

    // copy to device
    state_type x = x_host;
    state_type omega = omega_host;

    // create stepper
    runge_kutta4< state_type , value_type , state_type , value_type , thrust_algebra , thrust_operations > stepper;

    // create phase oscillator system function
    phase_oscillators sys( omega );

    // integrate
    integrate_const( stepper , sys , x , 0.0 , 10.0 , dt );

    thrust::copy( x.begin() , x.end() , std::ostream_iterator< value_type >( std::cout , "\n" ) );
    std::cout << std::endl;
    //]
}
